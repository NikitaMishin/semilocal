#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cmath>
#include "../memory_management.h"

using namespace cooperative_groups; // or...
using cooperative_groups::thread_group; // etc.


namespace bitwise_prefix_lcs_cuda {


    /**
     * @tparam T possible types are uint32 and uint64
     * @param a sum  of  two vectors
     * @param b  basically sum of two vectors plus one for each summator
     * @return  Update values for A and B and returns the  upper carry bit  for a value A
     */
    template<class T>
    inline   __device__ int kawanami_sum_reduction_without_if(T &a, T &b) {

        int with_no_carry = a > p; // carry bit of A
        int with_carry = max(with_no_carry, b > p); // carry bit of B


        T tmp; // tmp value of A
        int carry_tmp; // tmp value of carry bit of A
        int put_other_value; // weather or not we need to place a new value to current variable
        int active_mask;

#pragma  unroll
        for (int k = 1; k < 32; k <<= 1) {
            active_mask = (threadIdx.x % k) < k; // threads that on current iteration  can need to swap values

            // tmp values for carries and value
            carry_tmp = with_no_carry;
            tmp = a;

            // update A part
            put_other_value = __shfl_sync(0xffffffff, with_no_carry, k, k << 1); //broadcast
            put_b_value &= active_mask;


            a = (a & (put_other_value - 1)) | ((-put_other_value) & b);
            with_no_carry = (~put_other_value_value & with_no_carry) |
                            (put_other_value_value &
                             with_carry); // if put_b_value == 1 then we put with_no_carry <- with_carry;

            // update B part
            put_other_value = !__shfl_sync(0xffffffff, with_carry, k, k << 1);
            put_other_value &= active_mask;
            b = (tmp & (put_other_value - 1)) | ((-put_other_value) & b);
            with_carry = (~put_other_value & carry_tmp) | (put_other_value & with_carry);
        }

        return with_no_carry;
    }


    template<class T>
    inline   __device__ int kawanami_sum_reduction_with_if(T &a, T &b) {
        int with_no_carry = a > p;
        int with_carry = max(with_no_carry, b > p);

        T tmp;
        int carry_tmp;
        int put_other_value;
        int active_mask;

        #pragma  unroll
        for (int k = 1; k < 32; k <<= 1) {
            active_mask = (threadIdx.x % k) < k;

            carry_tmp = with_no_carry;
            tmp = a;

            // update A part
            put_other_value = __shfl_sync(0xffffffff, with_no_carry, k, k << 1);
            put_other_value &= active_mask;

            if (put_other_value) {
                a = b;
                with_no_carry = with_carry;
            }

            // update B part
            put_other_value = !__shfl_sync(0xffffffff, with_carry, k, k << 1);
            put_other_value &= active_mask;
            if (put_other_value) {
                b = tmp;
                with_carry = carry_tmp;
            }
        }

        return with_no_carry;
    }


    /**
     * With if
     * @tparam T
     * @param m
     * @param n_small | 32
     * @param a
     * @param lookup
     * @param vector_v
     * @param offset_x
     * @param offset_y
     * @param carries | 32
     */
    __global__ void hyrro_kawanami_kernel_without_shared(int m, int n_small, int *a  unsigned int *
    lookup, unsigned int *vector_v, int offset_x, int offset_y,
                                                         unsigned int *carries) {
        //TODO does it faster then with shared?

        // position relative to global
        int global_id_x = offset_x + 32 * blockIdx.x + threadIdx.x;
        int global_id_y = 1024 * (offset_y + blockId.x);
        int global_carry_id = 32 * (offset_y + blockId.x);


        //load packed carries from global memory
        unsigned int own_carry = carries[global_carry_id + threadIdx.x];

        unsigned int vector = vector_v[global_id_x];


        int loc = 0; // current position of packed carries for rows
        int bit_pos = 0; // current position of lower bit in loc
        unsigned int carry = 0; // either 1 or 0 for the lower bit of big vector number, for others always 0
        unsigned int carry_pack_to_save = 0; // packed carry bits to save, only upper carry bit is saved
        unsigned int processing_carry_pack = (threadIdx.x == 0) ? shared_carries[loc]
                                                                : 0; // only  lower adder can have carry

        #pragma unroll
        for (int i = 0; i < 1024; i++) {

            if (global_id_y > m) {
                // save partial
                // 31st broadcast carry_pack_to_save to all threads
                carry_pack_to_save = __shfl_sync(0xffffffff, carry_pack_to_save, 31, 32);
                if (threadIdx.x == loc) own_carry = carry_pack_to_save; // and loc lane will update own_carry

                break; // out of matrix
            }

            int key_a = a[global_id_y];
            T lookup_value = lookup[key_a * n_small + threadIdx.x];
            T p = vector & lookup_value;


            carry = ((1 << bit_pos) & processing_carry_pack) != 0;  // for others it would be always 0

            T sum_v = p + vector + carry;
            T sum_v_inc = sum_v + 1;

            carry_pack_to_save |= (kawanami_sum_reduction_without_if(sum_v, sum_v_inc) << bit_pos);
            vector = (vector ^ p) | sum_v; // update vector

            // if 31 then we need to save packed values and load another one
            if ((i % 32) == 31) {

                // 31st broadcast carry_pack_to_save to all threads
                carry_pack_to_save = __shfl_sync(0xffffffff, carry_pack_to_save, 31, 32);
                if (threadIdx.x == loc) own_carry = carry_pack_to_save; // and loc lane will update own_carry
                carry_pack_to_save = 0;

                loc++;
                // transfer carry pack from loc thread to  lane with id = loc
                processing_carry_pack = __shfl_sync(0xffffffff, own_carry, loc, 32);
                processing_carry_pack = (threadIdx.x == 0 && loc < 32) ? processing_carry_pack : 0;

                bit_pos = -1; // will be 0 at the end of this loop iteration
            }

            bit_pos++;
            global_id_y++;
        }

        // save 1024 bit vector back to memory
        vector_v[global_id_x] = vector;
        //save carries for the 1024 elements
        carries[global_carry_id + threadIdx.x] = own_carry;

    }


}


namespace bitwise_prefix_semi_local_lcs_cuda {


    /**
     * Given 64 unsigned number  0k_1...0k_32 convert it to k_1...k_2  32-bit unsigned int
     * Same idea could be applied to 00k_1....00k_m. Log based approach
     * @param n
     * @return
     */
    __device__ inline unsigned int calc_reduction(unsigned long long int n) {

        n &= (n >> 1) & (6148914691236517205ull);

        n |= (n >> 1);
        n &= (3689348814741910323ull);
        n |= ((n >> 2));
        n &= (1085102592571150095ull);
        n |= ((n >> 4));
        n &= (71777214294589695ull);
        n |= ((n >> 8));
        n &= (281470681808895ull);
        n |= ((n >> 16));
        n &= (4294967295ull);

        return unsigned int(n);
    }


    /**
     * Id function
     * @param n
     * @return
     */
    __device__ inline unsigned int calc_reduction(unsigned int n) {
        return n;
    }


    /**
     * Cell processing for binary and 4symbol
     *
     * @tparam T possible types are unsigned int (for binary), ull  for 4 symbol alphabet; for 16 symbol see paper
     * @tparam K possible values are 0 (unsigned int), 1 (unsigned long long)
     * @param l left packed 32 strands in machine word
     * @param t top packed 32 strands in machine word
     * @param a packed symbols of string a of type T (holds 32 symbols)
     * @param b packed symbols of string b of type T (holds 32 symbols)
     * @return updated values of l and t
     */
    template<class T, int K>
    __device__ inline void cell_processing(unsigned int &l, unsigned int &t, T a, T b) {

        unsigned int l_shifted = l;
        unsigned int t_shifted = t;
        unsigned int mask = 1;

        unsigned int cond;


#pragma unroll
        for (int shift = 31; shift > 0; shift--) {

            l_shifted = l >> shift;
            t_shifted = t << shift;

            cond = calc_reduction(~((a >> (shift << K)) ^ b));

            t = (l_shifted | (~mask)) & (t | (cond & mask));
            l = t_shifted ^ (t << shift) ^ l;

            mask = (mask << 1) | Input(1);
        }

        cond = calc_reduction(~(a ^ b));

        l_shifted = l;
        t_shifted = t;

        t = (l_shifted | (~mask)) & (t | (cond & mask));
        l = t_shifted ^ (t) ^ l;

        mask = unsigned int(-1);

#pragma unroll
        for (int shift = 1; shift < 32; shift++) {
            mask <<= 1;

            l_shifted = l << shift;
            t_shifted = t >> shift;

            cond = calc_reduction(~(((a << (shift << K)) ^ b)));

            t = (l_shifted | (~mask)) & (t | (cond & mask));
            l = t_shifted ^ (t >> shift) ^ l;
        }
    }



    /**
     *
     * @tparam Width  number of processing cells per thread. Witdh of one is a special case of antidiagonal patttern
     * @tparam T
     * @tparam K
     * @param a_reversed
     * @param size_a
     * @param b
     * @param size_b
     * @param left_strands
     * @param top_strands
     * @param offset_b
     * @param offset_a
     */
    template<int Width, class T, int K, DimBlock>
    __global__ void
    bitwise_prefix_semi_local_kernel(T *a_reversed, int size_a, T *b, int size_b,
                  unsigned int *left_strands, unsigned int *top_strands, int offset_b, int offset_a) {

        int per_warp = (32 - 1) + Width;        // per warp
        int per_block = per_warp * DimBlock;    // per block processed


        //todo dynamic shared
        volatile __shared__ b_part T[per_block];
        volatile __shared__ t_part unsigned int[per_block];

        auto lane_id = threadIdx.x % 32;
        auto warp_id = threadIdx.x / 32;


        auto global_id_col = offset_a +  lane_id + warp_id  * per_warp +   blockIdx.x * per_block;

        auto global_id_zero_in_block_col = offset_a + blockIdx.x * per_block + threadId.x;
        auto global_id_row = offset_b + threadIdx.x + blockIdx.x * blockDim.x;

        unsigned int l_strand = 0;
        unsigned int t_strand;

        T symbol_a = 0;
        T symbol_b;


        if (global_id_row >= 0 & global_id_row < size_a) {
            symbol_a = a_reversed[global_id_row];
            l_strand = left_strands[global_id_row];
        }

        #pragma unroll
        for (int i = 0; i < (per_block + DimBlock - 1) / DimBlock); i++) {
            auto glob_pos = global_id_zero_in_block_col + i * DimBlock;
            // todo i guess it is always >= 0 or not
            if ( (glob_pos >= 0) & ( glob_pos < size_b) ) {
                t_part[threadIdx.x+ i * DimBlock] = top_strands[glob_pos];
                b_part[threadIdx.x+ i * DimBlock] = b_part[glob_pos];
            }
        }

        int ptr_shared = warp_id * per_warp + lane_id;
        int global_ptr_shared = global_id_col;


        #pragma unroll
        for (int i = 0; i < Width; i++) {

            // todo is last condition really necessary?
            if (global_ptr_shared > 0 & global_ptr_shared < size_b & global_id_row > 0) {

                t_strand = t_part[ptr_shared];
                symbol_b = b_part[ptr_shared];

                cell_processing<T, K>(l_strand, t_strand, symbol_a, symbol_b);

                //update t_part
                t_part[ptr_shared] = t_part;
            }

            ptr_shared++;
            global_ptr_shared++;
        }


        // store l
        if (global_id_row >= 0 & global_id_row < size_a) left_strands[global_id_row] = l_strand;


        #pragma unroll
        for (int i = 0; i < (per_block + DimBlock - 1) / DimBlock); i++) {
            auto glob_pos = global_id_zero_in_block_col + i * DimBlock;
            if ( (glob_pos >= 0) & ( glob_pos < size_b) ) top_strands[glob_pos] t_part[threadIdx.x+ i * DimBlock];
        }

    }

}

/**
 * Fill array in pos with specified value for each cell
 * @tparam Input
 * @param arr
 * @param value
 * @param pos
 */
template<class Input>
__device__ inline void prefix_braid_init(Input *arr, Input value, int pos) {
    arr[pos] = value;
}


template<class Input>
__device__ inline void process_cube_withoutif(Input &symbol_a, Input &left_strand, Input &symbol_b, Input &top_strand,
                                              Input &l_active_mask, Input &r_active_mask, bool &use_with_mask,
                                              Input &braid_ones) {

    Input left_cap, symbols, combing_condition, rev_combing_cond, top_strand_shifted;

    Input mask = Input(1);

    // upper half
#pragma unroll
    for (int rev_counter = (sizeof(Input) * 8 - 2); rev_counter > 0; rev_counter -= 2) {
        left_cap = left_strand >> rev_counter;
        symbols = ~(((symbol_a >> rev_counter)) ^ symbol_b);
        symbols &= (symbols >> 1) & braid_ones;
        combing_condition = mask & (symbols | (((~(left_cap)) & top_strand)));

        if (use_with_mask) {
            combing_condition &= (l_active_mask >> rev_counter) & r_active_mask;
        }

        rev_combing_cond = combing_condition ^ braid_ones;

        top_strand_shifted = top_strand << rev_counter;
        top_strand = (rev_combing_cond & top_strand) | (combing_condition & left_cap);

        combing_condition <<= rev_counter;
        rev_combing_cond = combing_condition ^ braid_ones;

        left_strand = (rev_combing_cond & left_strand) | (combing_condition & top_strand_shifted);

        mask = (mask << 2) | Input(1);
    }


    // center
    symbols = (~(symbol_a ^ symbol_b));
    symbols &= (symbols >> 1) & braid_ones;
    combing_condition = (symbols | ((~left_strand) & top_strand));

    if (use_with_mask) {
        combing_condition &= (l_active_mask) & r_active_mask;
    }


    rev_combing_cond = combing_condition ^ braid_ones;

    top_strand_shifted = top_strand;
    top_strand = (rev_combing_cond & top_strand) | (combing_condition & left_strand);
    left_strand = (rev_combing_cond & left_strand) | (combing_condition & top_strand_shifted);

#pragma unroll
    for (int inside_diag_num = 2; inside_diag_num < (sizeof(Input) * 8 / 2 - 1) * 2 + 1; inside_diag_num += 2) {
        mask <<= 2;

        left_cap = left_strand << ((inside_diag_num));
        symbols = ~(((symbol_a << ((inside_diag_num)))) ^ symbol_b);
        symbols &= (symbols >> 1) & braid_ones;

        combing_condition = mask & (symbols | (((~(left_cap)) & top_strand)));

        if (use_with_mask) {
            combing_condition &= (l_active_mask << ((inside_diag_num))) & r_active_mask;
        }


        rev_combing_cond = combing_condition ^ braid_ones;

        top_strand_shifted = top_strand >> ((inside_diag_num));
        top_strand = (rev_combing_cond & top_strand) | (combing_condition & left_cap);

        combing_condition >>= ((inside_diag_num));
        rev_combing_cond = combing_condition ^ braid_ones;

        left_strand = (rev_combing_cond & left_strand) | (combing_condition & top_strand_shifted);
    }

}


template<class Input>
__global__ void prefix_braid_withoutif_prestored_lefts(
        Input const *seq_a_rev, int a_size, Input const *seq_b, int b_size,
        Input braid_one,
        Input *bitset_left_strand, Input *bitset_top_strand, int cells_per_thread_l, int cells_per_thread_t,
        int total_thds) {

    int num_diag = a_size + b_size - 1;
    int total_same_length_diag = num_diag - (a_size - 1) - (a_size - 1);
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    //sync primitive to sync whole grid
//    auto g = this_grid();
    auto g = this_thread_block();

    //init_phase
    for (int i = 0; i < cells_per_thread_l; i++) {
        if (total_thds * i + thread_id < a_size)
            prefix_braid_init(bitset_left_strand, braid_one, thread_id + total_thds * i);
    }

    for (int i = 0; i < cells_per_thread_t; i++) {
        if (total_thds * i + thread_id < a_size)
            prefix_braid_init(bitset_top_strand, Input(0), thread_id + total_thds * i);
    }


    Input left_strand = braid_one;
    Input left_symbol = (a_size - 1 - thread_id) >= 0 ? seq_a_rev[thread_id] : 0;
    bool use_with_mask = false;
//
    g.sync();

    //1 phase
    int b_pos = 0;
    for (int i = a_size - 1; i > 0; i--) {
        // only specific threads  && only active thread should perform
        if (thread_id >= i && thread_id < a_size) {
            Input symbol_b = seq_b[b_pos];
            Input top_strand = bitset_top_strand[b_pos];
            process_cube_withoutif(left_symbol, left_strand, symbol_b, top_strand, braid_one, braid_one,
                                   use_with_mask,
                                   braid_one);
            bitset_top_strand[b_pos] = top_strand;
            b_pos++;
        }
        g.sync();
    }


    //2 phase
    for (int i = 0; i < total_same_length_diag; i++) {
        if (thread_id < a_size) {
            Input symbol_b = seq_b[b_pos];
            Input top_strand = bitset_top_strand[b_pos];
            process_cube_withoutif(left_symbol, left_strand, symbol_b, top_strand, braid_one, braid_one, use_with_mask,
                                   braid_one);
            bitset_top_strand[b_pos] = top_strand;
            b_pos++;
        }
        g.sync();
    }

    //3 phase
    for (int i = a_size - 2; i >= 0; i--) {
        if (thread_id <= i) {
            Input symbol_b = seq_b[b_pos];
            Input top_strand = bitset_top_strand[b_pos];
            process_cube_withoutif(left_symbol, left_strand, symbol_b, top_strand, braid_one, braid_one, use_with_mask,
                                   braid_one);
            bitset_top_strand[b_pos] = top_strand;
            b_pos++;
        }
        g.sync();
    }

    bitset_left_strand[thread_id] = left_strand;
}


template<class Input>
__global__ void
process_diag(
        Input *sticky_braid, Input const *seq_a, int a_size, Input const *seq_b, int b_size, int offset_l,
        Input l_active_mAK,
        int offset_top, int diag_len) {
//
//    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
//    if (thread_id < diag_len) {
//
//
//        //load left symbol
//        Input a_symbol = seq_a[a_size - offset_l - 1 - thread_id];
//        Input left_strand = seq_a[thread_id + offset_l];
//        Input top_strand = sticky_braid[thread_id + offset_top + a_size];
//        Input b_symbol = seq_b[thread_id + offset_top];
//        bool use_with_mask =
//
//        process_cube_withoutif(a_symbol,left_strand,b_symbol,top_strand,)
//
//
//        sticky_braid[thread_id + offset_l] = (1 - should_swap) * left_strand + should_swap * top_strand;
//        sticky_braid[thread_id + offset_top + a_size] = (1 - should_swap) * top_strand + should_swap * left_strand;
//
//    }


}


template<class Input>
void four_symbol_gpu_runner_fully_gpu(Input *a_reverse_gpu, int a_size, int a_total_symbols,
                                      Input *b_gpu, int b_size, int b_total_symbols,
                                      Input *bitset_left_strand_gpu,
                                      Input *bitset_top_strand_gpu,
                                      int block_size) {
    Input braid_ones = Input(1);


    for (int shift = 0; shift < sizeof(Input) * 8 / 2; shift++) {
        braid_ones |= (braid_ones << shift * 2);
    }


    int cells_per_thd_l = 1;
    int cells_per_thd_t = std::ceil((1.0 * b_size) / a_size);
    int total_thds = a_size;
    dim3 grid(std::ceil(1.0 * a_size / block_size), 1);
    dim3 block(block_size, 1);
    prefix_braid_withoutif_prestored_lefts <<< grid, block >>>
            (a_reverse_gpu, a_size, b_gpu, b_size, braid_ones, bitset_left_strand_gpu, bitset_top_strand_gpu,
             cells_per_thd_l, cells_per_thd_t, total_thds);
    memory_management::gpuAssert(hipGetLastError(), __FILE__, __LINE__);

    memory_management::synchronize_with_gpu();

//    return bitset_left_strand_gpu;
}



//
//
//template<class Input>
//Input *four_symbol_fill_gpu_line(Input *a_reverse_gpu, int a_size, int a_total_symbols,
//                                        Input *b_gpu, int b_size, int b_total_symbols,
//                                        Input *bitset_left_strand_gpu,
//                                        Input *bitset_top_strand_gpu,
//                                        int block_size) {
//    Input braid_ones = Input(1);
//
//    for (int shift = 0; shift < sizeof(Input) * 8 / 2; shift++) {
//        braid_ones |= (braid_ones << shift * 2);
//    }
//
//    int cells_per_thd_l = 1;
//    int cells_per_thd_t = std::ceil((1.0 * b_size) / a_size);
//    int total_thds = a_size;
//    dim3 grid(std::ceil(1.0 * a_size / block_size), 1);
//    dim3 block(block_size, 1);
//
//    prefix_braid_withoutif_prestored_lefts<Input> <<< grid, block >>>
//            (a_reverse_gpu, a_size, b_gpu, b_size, braid_ones, bitset_left_strand_gpu, bitset_top_strand_gpu,
//             cells_per_thd_l, cells_per_thd_t, total_thds);
//
//    memory_management::gpuAssert(hipGetLastError(), __FILE__, __LINE__);
//
//    memory_management::synchronize_with_gpu();
//
//
//    return bitset_left_strand_gpu;
//}
//



// total thds = a_size




